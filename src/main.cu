#include "hip/hip_runtime.h"
﻿/*regressions*/

#include "koeffs.h"

double calculate_middle_time(
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator normal_iterator,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator normalIterator, unsigned long i1);

double calculate_coefficient_D(
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator normal_iterator,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator normalIterator,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator iterator1,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator iterator2,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator iterator3, double d,
        double d1);

void printf_eps(string file_name, vector<double> e) {
    int i;
    ofstream fout("out/" + file_name, ios_base::out);
    for (i = 0; i < e.size(); i++) {
        fout << fabs(e[i]) << "\n";
    }
    fout.close();
}

int main() {
    bool    init = 1, // если данные считаны корректно, то продолжаем работу программы, иначе делаем =0 и выходим
            flag = 1,    // корректность чтения лексемы из файла
            ex1 = 1, ex2 = 1, ex3 = 1, ex4 = 1;
    int i            //счетчик
    ;
    unsigned long             //счетчик
            N; // размерность массивов
    string Line1, Line2, Line3, Line4, Line5, Line; // строковые переменные для считывания слова из файла
    double Di,Ai, Bi, j = 0.1, S = 0, S0 = 0, a = 1, tau = 1,
            chis = 0, znam = 0, // промежуточные значения
            A = 0; //коэффициенты в уравнении регрессии

    vector<double> q_oil;    //добыча жидкости
    vector<double> q_water;    //добыча воды
    vector<double> times;    //время работы по месяцам
    vector<double> e; // массив со значениями невязок

    fstream f("data/db.txt");
    ofstream fout("out/result.txt", ios_base::out);

    // считывание входных данных из файла
    //// открытие файла

    if (f.is_open()) {
        while (flag) // в случае ошибки чтения flag = 0
        {

            if (!getline(f, Line1, '\t')) flag = 0;
            if (!getline(f, Line2, '\t')) flag = 0;
            if (!getline(f, Line3, '\t')) flag = 0;
            if (!getline(f, Line4, '\t')) flag = 0;
            if (!getline(f, Line5)) flag = 0;

            times.insert(times.end(), stof(Line3));
            q_oil.insert(q_oil.end(), stof(Line4));
            q_water.insert(q_water.end(), stof(Line5));
        }
        f.close();
    } else {
        cout << "Database not found or corrupted" << endl;
        init = 0;
    }

    if (init) // если файл открылся и данные считаны, то работаем дальше
    {

        // для расчета данных по нефти надо удалить нулевые значения.

        //проверка на некорректные значения: если значение добычи <=0 то удаляем его и соответствующее
        //ему значение времени работы скважины
        for (i = 0; i < q_oil.size(); i++) {
            if (q_oil[i] <= 0) {
                q_oil.erase(q_oil.begin() + i);
                times.erase(times.begin() + i);
                q_water.erase(q_water.begin() + i);
                if (i != 0) i--;
            }
        }
        N = q_oil.size();

        // выделение памяти под массивы на видеокарте
        // ЗАМЕЧАНИЕ при таком объявлении деструктор уже встроен, удалять в конце не надо
        thrust::device_vector<double> dev_tim(N);//время
        thrust::device_vector<double> dev_oil(N);//добыча нефти
        thrust::device_vector<double> dev_res(N);//массив для записи результата
        thrust::device_vector<double> dev_wat(N);//добыча воды
        thrust::device_vector<double> dev_buf(N);//для промежуточных результатов

        // вычисляем логарифмы величин дебита и заполняем массивы на девайсе (видеокарте) - с префиксом dev_
        for (i = 0; i < N; i++) {
            dev_oil[i] = log(q_oil[i]);
            dev_wat[i] = log(q_water[i]);
            dev_tim[i] = times[i];
        }

        // Вычисление сред значений: суммируем и делим на кол-во членов в сумме
        double middle_lnq_oil = thrust::reduce(dev_oil.begin(), dev_oil.end()) / N;
        double middle_time = calculate_middle_time(dev_tim.begin(),dev_tim.end(), N);

        // линейная регрессия имеет вид: y=A-Dt;

        double coefficient_D = calculate_coefficient_D(dev_tim.begin(),
                                                       dev_tim.end(),
                                                       dev_res.begin(),
                                                       dev_res.end(),
                                                       dev_oil.begin(), middle_time, middle_lnq_oil);
        A = middle_lnq_oil - coefficient_D * middle_time;

        // Вывод на экран и сохранение в файл
        cout << "Модель 1 для нефти: " << "f(t) = exp(- " << coefficient_D << "* t) \n" << endl;
        fout << "Модель 1 для нефти: " << "f(t) = exp(- " << coefficient_D << "* t) \n" << endl;

        //файл с невязками
        for (i = 0; i < N; i++) {
            e.insert(e.end(), A - coefficient_D * times[i] - q_oil[i]);
        }
        printf_eps("eps_O1.txt", e);

        // Модель 1 для воды.
        double middle_lnq_water = thrust::reduce(dev_wat.begin(), dev_wat.end()) / N;

        //по МНК найдем D:
        thrust::transform(dev_tim.begin(), dev_tim.end(), dev_wat.begin(), dev_res.begin(), num(middle_time, middle_lnq_water));
        chis = thrust::reduce(dev_res.begin(), dev_res.end());
        coefficient_D = chis / znam;
        A = middle_lnq_water - coefficient_D * middle_time;

        // вывод результата
        cout << "Модель 1 для жидкости: " << "f(t) = exp(- " << coefficient_D << "* t) \n" << endl;
        fout << "Модель 1 для жидкости: " << "f(t) = exp(- " << coefficient_D << "* t) \n" << endl;

        // создание файла невязок
        for (i = 0; i < N; i++) {
            e[i] = A - coefficient_D * times[i] - q_water[i];
        }

        printf_eps("eps_W1.txt", e);

        // MODEL 2
        // все как в модели 1 для воды. В уравнении регрессии замена ln(1+t) = z, но на коэффициенты это не влияет.

        cout << "Модель 2 для жидкости: " << "f(t) = (1 + t) ^ - " << coefficient_D << "\n" << endl;
        fout << "Модель 2 для жидкости: " << "f(t) = (1 + t) ^ - " << coefficient_D << "\n" << endl;

        printf_eps("eps_W2.txt", e);

        ////// MODEL 3
        // коэффиц-ты для лин регрессии уже были получены в модели 1. Наша задача - найти третий коэф-т, минимизируя невязки.

        Ai = 0.0001;
        Bi = 0.0001;
        Di = 100;    //нач знач для суммы S0

        thrust::transform(dev_oil.begin(), dev_oil.end(), dev_buf.begin(), findlog(Bi, Di));
        thrust::transform(dev_tim.begin(), dev_tim.end(), dev_buf.begin(), dev_res.begin(), findS(Ai, Bi));
        S0 = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);

        // Покоординатный спуск с целью поиска оптимального значения трех неизвестных коэффициентов
        while (ex1 || ex2 || ex3) {
            Di += j;

            thrust::transform(dev_oil.begin(), dev_oil.end(), dev_buf.begin(), findlog(Bi, Di));
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_buf.begin(), dev_res.begin(), findS(Ai, Bi));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);

            if (S0 - S >= 0) {
                S0 = S;
            } else {
                Di -= j;
                ex1 = 0;
            }

            // вторая координата
            Ai += j;
            thrust::transform(dev_oil.begin(), dev_oil.end(), dev_buf.begin(), findlog(Bi, Di));
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_buf.begin(), dev_res.begin(), findS(Ai, Bi));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);

            if (S - S0 <= 0) {
                S0 = S;
            } else {
                Ai -= j;
                ex2 = 0;
            }

            //третья координата
            Bi += j;
            thrust::transform(dev_oil.begin(), dev_oil.end(), dev_buf.begin(), findlog(Bi, Di));
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_buf.begin(), dev_res.begin(), findS(Ai, Bi));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);
            if (S - S0 <= 0) {
                S0 = S;
            } else {
                Bi -= j;
                ex3 = 0;
            }
        }
        // вывод резульата
        cout << "Модель 3 для нефти: " << "f(t) = (1 + " << Bi * Di << "* t) ^ (-" << /*1. / */Bi << ") \n" << endl;
        fout << "Модель 3 для нефти: " << "f(t) = (1 + " << Bi * Di << "* t) ^ (-" << /*1. / */Bi << ") \n" << endl;

        // подсчет и вывод невязок
        for (i = 0; i < N; i++) {
            e[i] = A - coefficient_D * times[i] - q_oil[i];
        }
        printf_eps("eps_O3.txt", e);

        //// MODEL 4
        i = 0;//ограничение числа итераций: 2000
        Ai = 0.01;
        Bi = 0.01;
        Di = 0.1;
        ex1 = 1;
        ex2 = 1;
        ex3 = 1;
        thrust::transform(dev_tim.begin(), dev_tim.end(), dev_oil.begin(), dev_res.begin(), findS4(Ai, Bi, Di, tau, a));
        S0 = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);

        // покоординатный спуск
        while ((ex1 || ex2 || ex3 || ex4) && i < 2000) {
            tau++;
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_oil.begin(), dev_res.begin(),
                              findS4(Ai, Bi, Di, tau, a));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);

            if (S - S0 <= 0) {
                S0 = S;
            } else {
                tau--;
                ex1 = 0;
            }

            a += j;
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_oil.begin(), dev_res.begin(),
                              findS4(Ai, Bi, Di, tau, a));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);
            if (S - S0 <= 0) {
                S0 = S;
            } else {
                a -= j;
                ex2 = 0;
            }

            //B
            Bi += j;
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_oil.begin(), dev_res.begin(),
                              findS4(Ai, Bi, Di, tau, a));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);
            if (S - S0 <= 0) {
                S0 = S;
            } else {
                Bi -= j;
                ex3 = 0;
            }

            //A
            Ai += j;
            thrust::transform(dev_tim.begin(), dev_tim.end(), dev_oil.begin(), dev_res.begin(),
                              findS4(Ai, Bi, Di, tau, a));
            S = thrust::reduce(dev_res.begin(), dev_res.end()) / (N - 2);
            if (S - S0 <= 0) {
                S0 = S;
            } else {
                Ai -= j;
                ex4 = 0;
            }
            i++;

        }        // end of while loop

        // вывод результата
        cout << "Модель 4 для нефти: " << "f(t) = (1 + " << Bi * Di << "* t) ^ (-" << 1. / Bi << ") \n" << endl;
        fout << "Модель 4 для нефти: " << "f(t) = (1 + " << Bi * Di << "* t) ^ (-" << 1. / Bi << ") \n"
             << endl;    // и в файл
        //невязки
        for (i = 0; i < N; i++) {
            e[i] = A - coefficient_D * times[i] - q_oil[i];
        }
        printf_eps("eps_O4.txt", e);

        e.clear();
        fout.close();
    }

    times.clear();
    q_oil.clear();
    q_water.clear();

    return 0;
}
/*
 * double coefficient_D = calculate_coefficient_D(dev_tim.begin(),
                                                       dev_tim.end(),
                                                       dev_res.begin(),
                                                       dev_res.end(),
                                                       dev_oil.begin(), middle_time, middle_lnq_oil);
 */

double calculate_coefficient_D(
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator dev_tim_begin,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator dev_tim_end,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator dev_res_begin,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator dev_res_end,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator dev_oil_begin,
        double middle_time,
        double middle_lnq_oil) {
    //по методу наименьших квадратов найдем D:
    thrust::transform(dev_tim_begin, dev_tim_end, dev_res_begin, den(middle_time));
    double znam = thrust::reduce(dev_res_begin, dev_res_end);
    thrust::transform(dev_tim_begin, dev_tim_end, dev_oil_begin, dev_res_begin, num(middle_time, middle_lnq_oil));
    double chis = thrust::reduce(dev_res_begin, dev_res_end);
    return chis / znam;
}

double calculate_middle_time(
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator begin,
        thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator end, unsigned long N) {
        return thrust::reduce(begin, end) / N;;
}